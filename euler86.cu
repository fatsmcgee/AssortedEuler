#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <vector>
#include <algorithm>
#include <sstream>

using namespace std;

// handy error macro:
#define GPU_CHECKERROR( err ) (gpuCheckError( err, __FILE__, __LINE__ ))
static void gpuCheckError(hipError_t err,
	const char *file,
	int line) {
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err),
			file, line);
	}
}

__global__ void find_cuboids(int * count, int *squares, int M)
{
	int a = blockIdx.x + 1;
	int b = blockIdx.y + 1;
	int c = blockIdx.z*blockDim.x + threadIdx.x + 1;

	if (c > M){
		return;
	}

	//only consider unique cuboids
	if (b > c || a > b){
		return;
	}

	bool hasSolution = false;

	for (int i = 0; i < 6; i++){

		//try every different rotation of the cuboid
		int w, l, h;

		if (i == 0){
			w = a; l = b; h = c;
		}
		else if (i == 1){
			w = a; h = b; l = c;
		}
		else if (i == 2){
			l = a; w = b; h = c;
		}
		else if (i == 3){
			l = a; h = b; w = c;
		}
		else if (i == 4){
			h = a; l = b; w = c;
		}
		else if (i == 5){
			h = a; w = b; l = c;
		}

		int solutionA = (w + h)*(w + h) + l*l;
		int solutionB = w*w + (l + h)*(l + h);
		int solution = solutionA < solutionB ? solutionA : solutionB;

		int solutionC = h*h + (l + w)*(l + w);
		solution = solutionC < solution ? solutionC : solution;

		int temp_i;
		if (squares[solution] == 1){
			hasSolution = true;
			break;
		}
	}

	if (hasSolution){
		atomicAdd(count, 1);
	}
}

int main(int argc, char ** argv)
{

	int M = 1000;
	stringstream s(argv[1]);
	s >> M;

	vector<int> h_Squares(100000000);
	
	//truth table for squares
	for (int i = 0; i < sqrt(h_Squares.size()); i++){
		h_Squares[i*i] = 1;
	}

	int * d_Squares;
	hipMalloc((void **)&d_Squares, h_Squares.size()*sizeof(int));
	hipMemcpy(d_Squares, &h_Squares[0],h_Squares.size()*sizeof(int), hipMemcpyHostToDevice);

	int *d_Count;
	hipMalloc((void**)&d_Count, sizeof(int));
	hipMemset(d_Count, 0, sizeof(int));

	dim3 gridDim(M, M, 2);
	dim3 blockDim(ceil(M/2)+1, 1, 1);

	find_cuboids << <gridDim, blockDim >> >(d_Count, d_Squares, M);

	int h_Count;
	hipMemcpy(&h_Count, d_Count, sizeof(int), hipMemcpyDeviceToHost);
	
	printf("%d\n", h_Count);

    return 0;
}

